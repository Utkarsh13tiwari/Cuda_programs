#include "hip/hip_runtime.h"
﻿#include<iostream>
#include<cstdlib>
#include<assert.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "time.h"

using namespace std;

#define BLOCK_SIZE 16

__global__ void matmul(int *a, int *b, int *c, int n) {

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;


	if (row < n && col < n) {
	int temp=0;
		for (int i = 0; i < n; i++) {
		// Matrices are stored in row-major order:
		// M(row, col) = *(M.elements + row * M.width + col)
			temp += a[row * n + i] * b[i * n + col];
		}
		c[row * n + col] = temp;
	}
}

__host__ void verify(int* a, int* b, int* c, int n) {

	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
		int temp=0;
			for (int k = 0; k < n; ++k)
			{
				temp += a[i * n + k] * b[k * n + j];
			}
			c[i * n + j] = temp;
		}
	}
}

int main() {
	int N = 1 << 10; // left shift to 10 place.
	size_t bytes = N * N * sizeof(int);

	int *a, *b, *c,*h_c;
	int* d_a, * d_b, * d_c;

	hipHostMalloc(&a, bytes);
	hipHostMalloc(&b, bytes);
	hipHostMalloc(&c, bytes);
	hipHostMalloc(&h_c, bytes);

	for (int i=0; i < N * N; ++i) {
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	float gpu_elapsed_time_ms;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// start to count execution time of GPU version
	hipEventRecord(start, 0);


	hipMallocManaged(&d_a, bytes);
	hipMallocManaged(&d_b, bytes);
	hipMallocManaged(&d_c, bytes);

	hipMemcpy(d_a, a,bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b,bytes, hipMemcpyHostToDevice);

	unsigned int block_rows = N + BLOCK_SIZE  / BLOCK_SIZE;
	unsigned int block_cols = N + BLOCK_SIZE  / BLOCK_SIZE;
	dim3 blockDim(block_rows, block_cols);
	dim3 threadDim(BLOCK_SIZE, BLOCK_SIZE);

	//dim3 numthreads(N / blocksize.x, N / blocksize.y);

	printf("Number of threads per block: %d \n\n", threadDim.x * threadDim.y);

	matmul << <blockDim,threadDim >> > (d_a, d_b, d_c, N);
	
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

	printf("Time for matrix multiplication of %d * %d on GPU: %f ms.\n\n", N, N, gpu_elapsed_time_ms);


	clock_t begin = clock();

	verify(a, b, c, N);

	clock_t end = clock();
	double time_spent = (double)1000 * (end - begin) / CLOCKS_PER_SEC;

	printf("Time elapsed on matrix multiplication of %d x %d on CPU: %f ms.\n\n",N, N, time_spent);


	int all_ok = 1;
	for (int i = 0; i < N; ++i)
	{
		for (int j = 0; j < N; ++j)
		{
			//printf("[%d][%d]:%d == [%d][%d]:%d\n", i, j, c[i*N + j], i, j, h_c[i*N + j]);
			if (h_c[i * N + j] != c[i * N + j])
			{
				all_ok = 0;
			}
		}
		//printf("\n");
	}

	// roughly compute speedup
	if (all_ok)
	{
		printf("all results are correct!!!, speedup = %f\n", time_spent / gpu_elapsed_time_ms);
	}
	else
	{
		printf("incorrect results\n");
	}


	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);
}