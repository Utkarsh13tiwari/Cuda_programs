#include "hip/hip_runtime.h"
﻿#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"
#include ""

__global__ void vecadd(int* c, const int* b, const int* a) {

	int id = blockIdx.x*blockDim.x+threadIdx.x;

	c[id] = a[id] + b[id];
}

int main() {
	const int s = 5;
	const int a[5] = { 1,2,3,4,5 };
	const int b[5] = { 4,5,6,7,8 };
	int c[5]={0};

	int* d_a=0;
	int* d_b=0;
	int* d_c=0;

	hipMalloc((void**)&d_a, 5 * sizeof(int));
	hipMalloc((void**)&d_b, 5 * sizeof(int));
	hipMalloc((void**)&d_c, 5 * sizeof(int));

	hipMemcpy(d_a, a, 5 * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, 5 * sizeof(int),hipMemcpyHostToDevice);

	vecadd << <1, 5 >> > (d_c, d_b, d_a);

	hipMemcpy(c, d_c, 5 * sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<5;i++)
		printf("%d ", c[i]);
	

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

